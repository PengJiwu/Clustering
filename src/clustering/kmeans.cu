#include "hip/hip_runtime.h"
#include "kmeans.h"
#include <hip/hip_math_constants.h>
#include <iostream>

__global__
void gpu_assign_elements_2_clusters(const unsigned int n_e, const unsigned int n_c, const float * d_ex, const float * d_ey, const float * d_cx, const float * d_cy, std::size_t * d_c){
   for(std::size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n_e; i += blockDim.x * gridDim.x){ 
        float best_cost = HIP_INF_F;
        for(std::size_t j = 0; j < n_c; ++j){
            float distance = fabsf(d_cx[j]-d_ex[i]) + fabsf(d_cy[j]-d_ey[i]);
            if(distance < best_cost){
                best_cost = distance;
                d_c[i] = j;
            }
        }
    }
}

void gpu_print_device_info(){
    int dev_id;
    hipDeviceProp_t prop;
    hipGetDevice(&dev_id);
    hipGetDeviceProperties(&prop, dev_id);
    std::cout<<"Device Number: "<< dev_id <<std::endl;
    std::cout<<"Device name: "<< prop.name <<std::endl;
    std::cout<<"Memory Clock Rate (KHz): "<< prop.memoryClockRate <<std::endl;
    std::cout<<"Memory Bus Width (bits): "<< prop.memoryBusWidth <<std::endl;
    std::cout<<"Peak Memory Bandwidth (GB/s): "<< 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 <<std::endl;
    std::cout<<"MultiProcessor count: "<< prop.multiProcessorCount<<std::endl;
    std::cout<<"Max threads per MultiProcessor: "<< prop.maxThreadsPerMultiProcessor <<std::endl;
}

void gpu_update_clusters_centers(const unsigned int n_e, const unsigned int n_c, const float * h_ex, const float * h_ey, float * h_cx, float * h_cy, const std::size_t * h_c){
#pragma omp parallel for
    for(std::size_t i = 0; i < n_c; ++i){
        float x_c = 0, y_c = 0;
        unsigned int count = 0;
        for(std::size_t j = 0; j < n_e; j++){
            if(h_c[j] == i){
                ++count;
                x_c += h_ex[j];
                y_c += h_ey[j];
            }
        }
        if(count != 0){
            h_cx[i] = x_c/count;
            h_cy[i] = y_c/count;
        }
    }
}

void clustering::Kmeans::gpu_kmeans(unsigned int iterations, unsigned int n_blocks, unsigned int n_threads_per_block){
    float *h_ex, *h_ey, *h_cx, *h_cy,
          *d_ex, *d_ey, *d_cx, *d_cy;
    std::size_t *h_c, *d_c;
    unsigned int n_e = elements.size(), n_c = clusters.size();

    h_ex = (float*)malloc(n_e*sizeof(float));
    h_ey = (float*)malloc(n_e*sizeof(float));
    h_cx = (float*)malloc(n_c*sizeof(float));
    h_cy = (float*)malloc(n_c*sizeof(float));
    h_c = (std::size_t *)malloc(n_e*sizeof(std::size_t));

    for(std::size_t i = 0; i < n_e; ++i){
        h_ex[i] = elements.at(i).x();
        h_ey[i] = elements.at(i).y();
    }

    clear_all_clusters();

    for(std::size_t i = 0; i < n_c; ++i){
        h_cx[i] = clusters.at(i).x();
        h_cy[i] = clusters.at(i).y();
    }

    hipMalloc(&d_ex, n_e*sizeof(float));
    hipMalloc(&d_ey, n_e*sizeof(float));
    hipMalloc(&d_cx, n_c*sizeof(float));
    hipMalloc(&d_cy, n_c*sizeof(float));
    hipMalloc(&d_c, n_e*sizeof(std::size_t));

    hipMemcpy(d_ex, h_ex, n_e*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ey, h_ey, n_e*sizeof(float), hipMemcpyHostToDevice);

    for(unsigned int i = 0; i < iterations; ++i){
        hipMemcpy(d_cx, h_cx, n_c*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_cy, h_cy, n_c*sizeof(float), hipMemcpyHostToDevice);

        gpu_assign_elements_2_clusters<<<n_blocks, n_threads_per_block>>>(n_e, n_c, d_ex, d_ey, d_cx, d_cy, d_c);

        hipMemcpy(h_c, d_c, n_e*sizeof(std::size_t), hipMemcpyDeviceToHost);

        gpu_update_clusters_centers(n_e, n_c, h_ex, h_ey, h_cx, h_cy, h_c);
    }

    for(std::size_t i = 0; i < n_e; ++i){
        elements.at(i).cluster(h_c[i]);
        clusters.at(h_c[i]).insert_element(i);
    }

    for(std::size_t i = 0; i < n_c; ++i)
        clusters.at(i).update_center(h_cx[i], h_cy[i]);

    hipFree(d_ex);
    hipFree(d_ey);
    hipFree(d_cx);
    hipFree(d_cy);
    hipFree(d_c);
    hipFree(h_ex);
    hipFree(h_ey);
    hipFree(h_cx);
    hipFree(h_cy);
    hipFree(h_c);
}
