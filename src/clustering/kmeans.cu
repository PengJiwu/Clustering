#include "hip/hip_runtime.h"
#include "kmeans.h"
#include <hip/hip_math_constants.h>
#include <iostream>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void clustering::Kmeans::do_saxpy(){
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

__global__
void gpu_assign_elements_2_clusters(const unsigned int n_e, const unsigned int n_c, const float * d_ex, const float * d_ey, const float * d_cx, const float * d_cy, std::size_t * d_c){
    for(std::size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n_e; i += blockDim.x * gridDim.x){
        float best_cost = HIP_INF_F;
        std::size_t best_cluster;
        for(std::size_t j = 0; j < n_c; ++j){
            float distance = fabsf(d_cx[j]-d_ex[i]) + fabsf(d_cy[j]-d_ey[i]);
            if(distance < best_cost){
                best_cost = distance;
                best_cluster = j;
            }
        }
    __syncthreads();
    d_c[i] = best_cluster;
    }
}

void gpu_print_device_info(){
    int dev_id;
    hipDeviceProp_t prop;
    hipGetDevice(&dev_id);
    hipGetDeviceProperties(&prop, dev_id);
    std::cout<<"Device Number: "<< dev_id <<std::endl;
    std::cout<<"Device name: "<< prop.name <<std::endl;
    std::cout<<"Memory Clock Rate (KHz): "<< prop.memoryClockRate <<std::endl;
    std::cout<<"Memory Bus Width (bits): "<< prop.memoryBusWidth <<std::endl;
    std::cout<<"Peak Memory Bandwidth (GB/s): "<< 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 <<std::endl;
    std::cout<<"MultiProcessor count: "<< prop.multiProcessorCount<<std::endl;
    std::cout<<"Max threads per MultiProcessor: "<< prop.maxThreadsPerMultiProcessor <<std::endl;
}

void gpu_update_clusters_centers(const unsigned int n_e, const unsigned int n_c, const float * h_ex, const float * h_ey, float * h_cx, float * h_cy, const std::size_t * h_c){
#pragma omp parallel for
    for(std::size_t i = 0; i < n_c; ++i){
        float x_c = 0, y_c = 0;
        unsigned int count = 0;
        for(std::size_t j = 0; j < n_e; j++){
            if(h_c[j] == i){
                ++count;
                x_c += h_ex[j];
                y_c += h_ey[j];
            }
        }
        if(count != 0){
            h_cx[i] = x_c/count;
            h_cy[i] = y_c/count;
        }
    }
}

void clustering::Kmeans::gpu_kmeans(unsigned int iterations, unsigned int n_blocks, unsigned int n_threads_per_block){
    float *h_ex, *h_ey, *h_cx, *h_cy,
          *d_ex, *d_ey, *d_cx, *d_cy;
    std::size_t *h_c, *d_c;
    unsigned int n_e = elements.size(), n_c = clusters.size();

    h_ex = (float*)malloc(n_e*sizeof(float));
    h_ey = (float*)malloc(n_e*sizeof(float));
    h_cx = (float*)malloc(n_c*sizeof(float));
    h_cy = (float*)malloc(n_c*sizeof(float));
    h_c = (std::size_t *)malloc(n_e*sizeof(std::size_t));

    for(std::size_t i = 0; i < n_e; ++i){
        h_ex[i] = elements.at(i).x();
        h_ey[i] = elements.at(i).y();
    }

    clear_all_clusters();

    for(std::size_t i = 0; i < n_c; ++i){
        h_cx[i] = clusters.at(i).x();
        h_cy[i] = clusters.at(i).y();
    }

    std::chrono::high_resolution_clock::time_point time_start, time_end;
    time_start = std::chrono::high_resolution_clock::now();

    hipMalloc(&d_ex, n_e*sizeof(float));
    hipMalloc(&d_ey, n_e*sizeof(float));
    hipMalloc(&d_cx, n_c*sizeof(float));
    hipMalloc(&d_cy, n_c*sizeof(float));
    hipMalloc(&d_c, n_e*sizeof(std::size_t));

    hipMemcpy(d_ex, h_ex, n_e*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ey, h_ey, n_e*sizeof(float), hipMemcpyHostToDevice);

    for(unsigned int i = 0; i < iterations; ++i){
        hipMemcpy(d_cx, h_cx, n_c*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_cy, h_cy, n_c*sizeof(float), hipMemcpyHostToDevice);

        gpu_assign_elements_2_clusters<<<n_blocks, n_threads_per_block>>>(n_e, n_c, d_ex, d_ey, d_cx, d_cy, d_c);

        hipMemcpy(h_c, d_c, n_e*sizeof(std::size_t), hipMemcpyDeviceToHost);

        gpu_update_clusters_centers(n_e, n_c, h_ex, h_ey, h_cx, h_cy, h_c);
    }

    time_end = std::chrono::high_resolution_clock::now();
    auto total_time = time_end - time_start;
    std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(total_time).count()<<" ms "<<std::endl;

    for(std::size_t i = 0; i < n_e; ++i){
        elements.at(i).cluster(h_c[i]);
        clusters.at(h_c[i]).insert_element(i);
    }

    for(std::size_t i = 0; i < n_c; ++i)
        clusters.at(i).update_center(h_cx[i], h_cy[i]);

    hipFree(d_ex);
    hipFree(d_ey);
    hipFree(d_cx);
    hipFree(d_cy);
    hipFree(d_c);
    hipFree(h_ex);
    hipFree(h_ey);
    hipFree(h_cx);
    hipFree(h_cy);
    hipFree(h_c);
}
